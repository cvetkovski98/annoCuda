
#include <hip/hip_runtime.h>
extern "C"
__global__
void matrixMultiplicationKernel(double* A, double* B, double* C, long N) {

    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    double tmpSum = 0.0;

    if (ROW < N && COL < N) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < N; i++) {
            tmpSum += A[ROW * N + i] * B[i * N + COL];
        }
    }
    C[ROW * N + COL] = tmpSum;
}
